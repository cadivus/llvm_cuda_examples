#include "hip/hip_runtime.h"
#include <stdio.h>

#include <initializer_list>
#ifdef __clang__
#include <offload/cuda/hip/hip_runtime.h>
#endif

// CUDA kernel function: Each thread will execute this function
__global__ void helloKernel(int *A) {
    // Print from the GPU device.
    // Only one thread (thread index 0) will print to avoid duplicate messages.
    if (threadIdx.x == 0 && blockIdx.x == 0) {
	      *A = 42;
	      // Broken
        // printf("Hello, World from the GPU %i!\n", 0);
    }
}

int main() {
    int *A;
    int R = 5, P = 7;
    hipMalloc(&A, 4);
    hipMemcpy(A, &P, 4, hipMemcpyHostToDevice);
    // Launch the kernel with one block and one thread.
    helloKernel<<<1, 1>>>(A);

    // Wait for the GPU to finish before accessing on host
    hipDeviceSynchronize();

    hipMemcpy(&R, A, 4, hipMemcpyDeviceToHost);

    // Optionally, also print a message from the host
    printf("Hello, World from the CPU: %i!\n", R);

    return 0;
}
