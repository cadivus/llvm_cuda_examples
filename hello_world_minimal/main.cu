#include <stdio.h>

#include <initializer_list>
#ifdef __clang__
#include <offload/cuda/hip/hip_runtime.h>
#endif

int main() {
    int *A;
    int R = 5, P = 7;
    hipMalloc(&A, 4);
    hipMemcpy(A, &P, 4, hipMemcpyHostToDevice);

    hipMemcpy(&R, A, 4, hipMemcpyDeviceToHost);

    // Optionally, also print a message from the host
    printf("Hello, World from the CPU: %i!\n", R);

    return 0;
}
